﻿
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>
#include <chrono>


#define BLOCKS 1024 * 1024
#define THREADS 256


__global__ void FindKey(uint64_t aPlainText, uint64_t aKeyMax, uint64_t aCipherExpected, uint64_t* aResult)
{
    int index = blockIdx.x * THREADS + threadIdx.x;
    int stride = BLOCKS * THREADS;

    if (*aResult)
    {
        return;
    }

    for (uint64_t key = index; key < aKeyMax; key += stride)
    {
        if (*aResult)
        {
            return;
        }

        uint64_t aWord = aPlainText;
        uint64_t aKey = key;

        for (char roundIndex = 0; roundIndex < 20; roundIndex++)
        {
            aWord ^= aKey;

            uint64_t sbox[] = { 0x2, 0x4, 0x5, 0x6, 0x1, 0xA, 0xF, 0x3, 0xB, 0xE, 0x0, 0x7, 0x9, 0x8, 0xC, 0xD };

            uint64_t wordNew = 0;

            for (char nibbleIndex = 0; nibbleIndex < 16; nibbleIndex++)
            {
                char nibble = (aWord >> (nibbleIndex * 4)) & 0xF;

                wordNew |= sbox[nibble] << (nibbleIndex * 4);
            }

            aWord = wordNew;

            aWord = ((aWord << 15) | (aWord >> (64 - 15))) ^ ((aWord << 32) | (aWord >> (64 - 32))) ^ aWord;

            aKey = ((aKey << 15) | (aKey >> (64 - 15))) ^ ((aKey << 32) | (aKey >> (64 - 32))) ^ aKey ^ 0x3;
        }

        if (aWord == aCipherExpected)
        {
            *aResult = key;
            return;
        }
    }
}


int main()
{
    // Assignment to find the key for
    // 0 - 20 bit key
    // 1 - 32 bit key
    // 2 - 44 bit key, cunldn't find :(
    char testCase = 2;

    uint64_t* resultHost;
    resultHost = (uint64_t*)malloc(sizeof(uint64_t));
    *resultHost = false;

    uint64_t* resultDevice;
    hipMalloc((void**)&resultDevice, sizeof(uint64_t));
    hipMemcpy(resultDevice, resultHost, sizeof(uint64_t), hipMemcpyHostToDevice);

    // Find the key
    uint64_t plainTextVector[] =        { 0x0441E17A4283531C,   0x2E01B3D043EFA867, 0x176BF4819739A044  };
    uint64_t keyMaxVector[] =           { 0x0000000000100000,   0x0000000100000000, 0x0000100000000000  };
    uint64_t cipherExpectedVector[] =   { 0x4546788A5ADF2106,   0x839C3F5FD7CFA5E5, 0x3422F9DFE688E023  };

    auto start = std::chrono::high_resolution_clock::now();

    FindKey<<<BLOCKS, THREADS>>>(plainTextVector[testCase], keyMaxVector[testCase], cipherExpectedVector[testCase], resultDevice);

    hipMemcpy(resultHost, resultDevice, sizeof(uint64_t), hipMemcpyDeviceToHost);

    std::cout << "Key: " << std::hex << *resultHost << std::endl;

    std::cout << "RunTime: " << std::dec << std::chrono::duration_cast<std::chrono::seconds>(std::chrono::high_resolution_clock::now() - start).count() << "[seconds]\n";

    return 0;
}
